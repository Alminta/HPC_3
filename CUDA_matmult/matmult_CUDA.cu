#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "kernfuncs.h"
#include "hipblas.h"

extern"C"{

#include <cblas.h>
    
void matmult_lib(int m, int n, int k, double *A, double *B, double *C)
{
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1, A, k, B, n, 0, C, n);
}


void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    const double alpha=1.0,beta=0.0;

    double *d_A,*d_B,*d_C;

    hipMalloc( (void**)&d_A , m * k * sizeof(double));
    hipMalloc( (void**)&d_B , n * k * sizeof(double));
    hipMalloc( (void**)&d_C , m * n * sizeof(double));

    hipMemcpy(d_A,A,m*k*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,n*k*sizeof(double),hipMemcpyHostToDevice);

    hipblasDgemm(handle, transa, transb, m, n, k, &alpha, d_A, k, d_B, n, &beta, d_C, n);

    hipMemcpy(C,d_C,m*n*sizeof(double),hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C)
{
    
    double *d_A,*d_B,*d_C;
    
    hipMalloc( (void**)&d_A , m * k * sizeof(double));
    hipMalloc( (void**)&d_B , n * k * sizeof(double));
    hipMalloc( (void**)&d_C , m * n * sizeof(double));
    
    hipMemcpy(d_A,A,m*k*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,n*k*sizeof(double),hipMemcpyHostToDevice);

    matmult_gpu1_kern <<<1,1>>>(m,n,k,d_A,d_B,d_C);
    hipDeviceSynchronize();

    hipMemcpy(C,d_C,m*n*sizeof(double),hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C)
{
    int bsize = 32;
    double *d_A,*d_B,*d_C;
    
    hipMalloc( (void**)&d_A , m * k * sizeof(double));
    hipMalloc( (void**)&d_B , n * k * sizeof(double));
    hipMalloc( (void**)&d_C , m * n * sizeof(double));
    
    hipMemcpy(d_A,A,m*k*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,n*k*sizeof(double),hipMemcpyHostToDevice);

    dim3 dimGrid((n+bsize-1)/bsize,(m+bsize-1)/bsize);
    dim3 dimBlock(bsize,bsize);
    matmult_gpu2_kern <<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);
    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(C,d_C,m*n*sizeof(double),hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C)
{
    int bsize = 32,R = 2;
    double *d_A,*d_B,*d_C;
    
    hipMalloc( (void**)&d_A , m * k * sizeof(double));
    hipMalloc( (void**)&d_B , n * k * sizeof(double));
    hipMalloc( (void**)&d_C , m * n * sizeof(double));
    
    hipMemcpy(d_A,A,m*k*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,n*k*sizeof(double),hipMemcpyHostToDevice);

    dim3 dimGrid((n + bsize - 1)/bsize,((m+R-1)/R+bsize-1)/bsize);
    dim3 dimBlock(bsize,bsize);
    matmult_gpu3_kern <<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);
    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(C,d_C,m*n*sizeof(double),hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C)
{
    int bsize = 32, R=8;
    double *d_A,*d_B,*d_C;
    
    hipMalloc( (void**)&d_A , m * k * sizeof(double));
    hipMalloc( (void**)&d_B , n * k * sizeof(double));
    hipMalloc( (void**)&d_C , m * n * sizeof(double));
    
    hipMemcpy(d_A,A,m*k*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,n*k*sizeof(double),hipMemcpyHostToDevice);

    dim3 dimGrid((n + bsize - 1)/bsize,((m+R-1)/R+bsize-1)/bsize);
    dim3 dimBlock(bsize,bsize);
    matmult_gpu4_kern <<<dimGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);
    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(C,d_C,m*n*sizeof(double),hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

#ifndef __MATRIX
#define __MATRIX
typedef struct
{
    int width;
    int height;
    int stride;
    double *elements;
} Matrix;
#endif

#ifndef __BLOCKSIZE
#define BLOCK_SIZE 16
#endif


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(double);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
        hipMemcpyHostToDevice);
        Matrix d_B;
        d_B.width = d_B.stride = B.width; d_B.height = B.height;
        size = B.width * B.height * sizeof(double);
        hipMalloc(&d_B.elements, size);
        hipMemcpy(d_B.elements, B.elements, size,
            hipMemcpyHostToDevice);

            // Allocate C in device memory
            Matrix d_C;
            d_C.width = d_C.stride = C.width; d_C.height = C.height;
            size = C.width * C.height * sizeof(double);
            hipMalloc(&d_C.elements, size);
            
            // Invoke kernel
            dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
            dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
            MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
            checkCudaErrors(hipDeviceSynchronize());
            // Read C from device memory
            hipMemcpy(C.elements, d_C.elements, size,
                hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C){
    Matrix MA,MB,MC;
    MB.width=MC.width=n;
    MA.width=MB.height=k;
    MA.height=MC.height=m;
    MA.elements = A;
    MB.elements = B;
    MC.elements = C;
    MatMul(MA,MB,MC);

}

}
