#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "print.h"
#include "init.h"
#include "jacobi.h"

#define N_DEFAULT 100

int
main(int argc, char *argv[]) 
{

    int 	N = N_DEFAULT;
    int 	iter_max = 1000;
    double	tolerance=1000;
    double	start_T=0;
    int		output_type = 0;
    char	*output_prefix = "poisson_res";
    char    *output_ext    = "";
    char	output_filename[FILENAME_MAX];
    double 	*h_u1 = NULL;
    double 	*h_u2 = NULL;
    double 	*h_f = NULL;
    double 	*d_u1 = NULL;
    double 	*d_u2 = NULL;
    double 	*d_f = NULL;


    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    tolerance = atof(argv[3]);  // tolerance
    start_T   = atof(argv[4]);  // start T for all inner grid points
    if (argc == 6) {
	output_type = atoi(argv[5]);  // ouput type
    }
    
    
    long long total_size = N * N * N * sizeof(double);


    // allocate memory
    hipHostMalloc((void**)&h_u1, total_size);
    hipHostMalloc((void**)&h_u2, total_size);
    hipHostMalloc((void**)&h_f, total_size);
    hipMalloc((void**)&d_u1, total_size);
    hipMalloc((void**)&d_u2, total_size);
    hipMalloc((void**)&d_f, total_size);


    // check allocation
    if ( h_u1 == NULL ) {
        perror("array h_u1: allocation failed");
        exit(-1);
    }
    if ( h_u2 == NULL ) {
        perror("array h_u2: allocation failed");
        exit(-1);
    }
    if ( h_f == NULL ) {
        perror("array h_f: allocation failed");
        exit(-1);
    }
    if ( d_u1 == NULL ) {
        perror("array d_u1: allocation failed");
        exit(-1);
    }
    if ( d_u2 == NULL ) {
        perror("array d_u2: allocation failed");
        exit(-1);
    }
    if ( d_f == NULL ) {
        perror("array d_f: allocation failed");
        exit(-1);
    }


    // initialize arrays on host
    initialize(h_u1, h_u2, h_f, N, start_T);
    

    // copy arrays to GPU
    hipMemcpy(d_u1, h_u1, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_u2, h_u2, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_f, h_f, total_size, hipMemcpyHostToDevice);


    // calculate poisson problem with the jacobi method
    jacobi(d_u1, d_u2, d_f, N, iter_max, tolerance);


    // copy memory to CPU
    hipMemcpy(h_u1, d_u1, total_size, hipMemcpyDeviceToHost);
    

    // dump  results if wanted 
    switch(output_type) {
	case 0:
	    // no output at all
	    break;
	case 3:
	    output_ext = ".bin";
	    sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
	    fprintf(stderr, "Write binary dump to %s: ", output_filename);
	    print_binary(output_filename, N, h_u1);
	    break;
	case 4:
	    output_ext = ".vtk";
	    sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
	    fprintf(stderr, "Write VTK file to %s: ", output_filename);
	    print_vtk(output_filename, N, h_u1);
	    break;
	default:
	    fprintf(stderr, "Non-supported output type!\n");
	    break;
    }
    

    // de-allocate memory
    hipFree(h_u1);
    hipFree(h_u2);
    hipFree(h_f);
    hipFree(d_u1);
    hipFree(d_u2);
    hipFree(d_f);

    return(0);
}
